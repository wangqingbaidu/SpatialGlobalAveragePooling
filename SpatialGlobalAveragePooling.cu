#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"

template <typename Dtype>
__global__ void GlobalAvePoolForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {    
    const int pool_size = height * width;
    Dtype aveval = 0;
    const Dtype* const bottom_slice = bottom_data + index * height * width;
    for (int h = 0; h < height; ++h) {
      for (int w = 0; w < width; ++w) {
        aveval += bottom_slice[h * width + w];
      }
    }
   top_data[index] = aveval / pool_size;
  }
}


void THNN_CudaSpatialGlobalAveragePooling_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *output)
{
  THCUNN_assertSameGPU(state, 2, input, output);
  THArgCheck(input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch) tensor expected");

  long nInputCols, nInputRows, nInputPlane, batchSize;
  long nOutputCols = 1, nOutputRows = 1;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }
 
  input = THCudaTensor_newContiguous(state, input);
  float* input_data = THCudaTensor_data(state, input);

  THCudaTensor_resize4d(state, output, batchSize, nInputPlane, nOutputRows, nOutputCols);

  float* output_data = THCudaTensor_data(state, output);

  int count = THCudaTensor_nElement(state, output);

  GlobalAvePoolForward<float>
	  <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>(
		count, input_data, batchSize, nInputPlane, nInputRows, nInputCols, output_data);
  
  THCudaCheck(hipGetLastError());

  if(input->nDimension == 3)
    THCudaTensor_resize3d(state, output, nInputPlane, nOutputRows, nOutputCols);

  THCudaTensor_free(state, input);

}

template <typename Dtype>
__global__ void GlobalAvePoolBackward(const int nthreads, const Dtype* const top_diff,
    const int num, const int channels, const int height, const int width, Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
	const int pool_size = height * width;
    const Dtype* const top_diff_slice = top_diff + index / pool_size;
    Dtype gradient = top_diff_slice[0] / pool_size;
    bottom_diff[index] = gradient;
  }
}

void THNN_CudaSpatialGlobalAveragePooling_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *gradOutput, THCudaTensor *gradInput)
{
  THCUNN_assertSameGPU(state, 3, input, gradOutput, gradInput);

  input = THCudaTensor_newContiguous(state, input);
  gradOutput = THCudaTensor_newContiguous(state, gradOutput);

  long nInputCols, nInputRows, nInputPlane, batchSize;
//  long nOutputCols = 1, nOutputRows = 1;
  
  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  THCudaTensor_resizeAs(state, gradInput, input);

  int count = THCudaTensor_nElement(state, input);
  
    GlobalAvePoolBackward<float>
      <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
        (count,
        THCudaTensor_data(state, gradOutput),
        batchSize, nInputPlane, nInputRows, nInputCols,
        THCudaTensor_data(state, gradInput));
  
  THCudaCheck(hipGetLastError());

  // clean
  THCudaTensor_free(state, input);
  THCudaTensor_free(state, gradOutput);
}

